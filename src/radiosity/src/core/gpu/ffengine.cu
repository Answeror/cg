/**
 *  @file
 *  @author answeror <answeror@gmail.com>
 *  @date 2012-04-05
 *  
 *  @section DESCRIPTION
 *  
 *  
 */

#include "ffengine.hpp"
#include "ffengine_impl.hpp"

namespace gpu = cg::gpu;

namespace
{
    void go(
        int width,
        int height, 
        hipGraphicsResource_t cuda_resource,
        const float *coeffs, ///< device side
        float *ffs ///< host side, must be inited outside
        );
}

void gpu::ffengine_method::calc_ff()
{
    go(EDGE_2, EDGE_2, data->cuda_resource, 
}