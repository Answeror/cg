#include "hip/hip_runtime.h"
/**
 *  @file
 *  @author answeror <answeror@gmail.com>
 *  @date 2012-04-05
 *  
 *  @section DESCRIPTION
 *  
 *  
 */

#include <iostream>

#include "calcff.hpp"
#include "glcu.hpp"

namespace gpu = cg::gpu;

namespace
{
    const int EDGE_1 = 256;	 ///< size (in pixels) of hemi-cube edge
    const int EDGE_2 = 2*EDGE_1;	///< EDGE_1 * 2 (size of important area in hemicube)
    const int EDGE_LENGTH = 3*EDGE_1;	 ///< size (pixels) of render viewport

    texture<uchar4, hipTextureType2D, hipReadModeElementType> tex;

    __device__ inline unsigned int decode_color(unsigned int r, unsigned int g, unsigned int b)
    {
        return r + (g << 8) + (b << 16);
    }

    __device__ inline void atomicadd(float* address, float value){
#if __CUDA_ARCH__ >= 200 // for Fermi, atomicAdd supports floats
        atomicAdd(address,value);
#elif __CUDA_ARCH__ >= 110
        // float-atomic-add
        float old = value;
        while ((old = atomicExch(address, atomicExch(address, 0.0f)+old))!=0.0f);
#endif
    }

    __global__ void count_pixels(float *ffs, const float *coeffs)
    {
        auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
        //if (x >= 128 && x < 128 + 512 && y >= 128 && y < 128 + 512)
        {
            auto read = tex2D(tex, x + 128 + 0.5f, y + 128 + 0.5f);
            auto id = decode_color(read.x, read.y, read.z);
            if (id != 0xffffff)
            {
                atomicadd(&ffs[id], (float)coeffs[x * EDGE_2 + y]);
            }
        }
    }
}

void gpu::calcff(
    int patch_count,
    int width,
    int height, 
    hipGraphicsResource_t target,
    const float *coeffs, ///< device side
    float *ffs ///< host side
    )
{
    HANDLE_ERROR(hipGraphicsMapResources(1, &target, NULL));
    hipArray *arr;
    HANDLE_ERROR(hipGraphicsSubResourceGetMappedArray(&arr, target, 0, 0));
    HANDLE_ERROR(hipBindTextureToArray(tex, arr));

    const int DIM = EDGE_2;
    dim3 grids(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    float *dffs = nullptr;
    HANDLE_ERROR(hipMalloc((void**)&dffs, sizeof(float) * patch_count));
    HANDLE_ERROR(hipMemset((void*)dffs, 0, sizeof(float) * patch_count));
    count_pixels<<<grids, threads>>>(dffs, coeffs);

    HANDLE_ERROR(hipMemcpy(ffs, dffs, sizeof(float) * patch_count, hipMemcpyDeviceToHost));

#if 0
    {
        int n = 0;
        for (int i = 0; i < patch_count; ++i)
        {
            if (ffs[i] > 0) ++n;
        }
        std::cout << "ffs not zero: " << n << std::endl;
    }
#endif

    HANDLE_ERROR(hipGraphicsUnmapResources(1, &target, NULL));
    HANDLE_ERROR(hipUnbindTexture(tex));
}