#include "hip/hip_runtime.h"
/**
 *  @file
 *  @author answeror <answeror@gmail.com>
 *  @date 2012-03-26
 *  
 *  @section DESCRIPTION
 *  
 *  
 */

#include <iostream>
#include <fstream>
#include <cstdlib>

#include <GL/glew.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <hip/hip_texture_types.h>

#include "core/glcu.hpp"

void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

texture<uchar4, hipTextureType2D, hipReadModeElementType> tex;

__global__ void kernel(uchar4 *p)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    uchar4 read = tex2D(tex, x + 0.5f, y + 0.5f);
    p[y * 512 + x] = read;
}

void output(uchar4 *p, int W, int H)
{
    std::ofstream f("out.ppm");
    f << "P3\n" << int(W) << " " << int(H) << "\n255\n";
    for(int y=0; y<H; ++y )
    {
        for(int x=0; x<W; ++x )
        {
            unsigned char *color = (unsigned char*)(p + y * W + x);
            for( size_t c=0; c<3; ++c )
                f << int(color[c]) << " ";
        }
    }
}

void cg::glcu(hipGraphicsResource_t target, int width, int height)
{
    HANDLE_ERROR(hipGraphicsMapResources(1, &target, NULL));
    hipArray *arr;
    HANDLE_ERROR(hipGraphicsSubResourceGetMappedArray(&arr, target, 0, 0));
    HANDLE_ERROR(hipBindTextureToArray(tex, arr));

    const int DIM = 512;
    dim3 grids(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    uchar4 *p = nullptr;
    HANDLE_ERROR(hipMalloc((void**)&p, sizeof(uchar4) * width * height));
    kernel<<<grids, threads>>>(p);

    uchar4 q[512 * 512];
    HANDLE_ERROR(hipMemcpy(q, p, sizeof(uchar4) * width * height, hipMemcpyDeviceToHost));
    output(q, width, height);

    HANDLE_ERROR(hipGraphicsUnmapResources(1, &target, NULL));
    HANDLE_ERROR(hipUnbindTexture(tex));
    //hipFreeArray(arr);
    hipFree(p);
}